#include "hip/hip_runtime.h"
﻿/*
Author: Adithi Upadhya
Class: ECE6122
Last Date Modified: 11/08/2023
Description: CUDA-based 2D Random Walk Simulation
*/

#include <iostream>
#include <vector>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""


//#define NUM_BLOCKS 256
//#define THREADS_PER_BLOCK 256

__global__ void RandomWalk(uint64_t* position_x, uint64_t* position_y, uint64_t num_walkers, uint64_t num_steps, unsigned int seed)
{
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);
    if (tid < num_walkers)
    {
        int x = 0; 
        int y = 0;
        for (unsigned int i = 0; i < num_steps; ++i)
        {
            float randv = hiprand_uniform(&state); 
            if (randv < 0.25)
                x -= 1; //go left
            else if (randv >= 0.25 && randv < 0.5)
                x += 1; //go right
            else if (randv >= 0.5 && randv < 0.75)
                y += 1; //go up
            else
                y -= 1; //go down
        }
        position_x[tid] = x; 
        position_y[tid] = y; 
    }
}



int main(int argc, char* argv[])
{   
    uint64_t num_walkers, num_steps; 
    hipEvent_t startEvent, stopEvent; 
    float elapsed_time;

    //default values
    num_walkers = 10000;
    num_steps = 1000000;
    
    //flags to track options, argc contains no. of parameters
    for (int i = 1; i < argc; i += 2) 
    {
        if (argv[i][1] == 'W') 
        {
            if (!isdigit(argv[i + 1][0]))
            {
                std::cerr << "Invalid input" << std::endl;
                return 1;
            }
            else
                num_walkers = atoi(argv[i + 1]);
        }
        else if (argv[i][1] == 'I') {
            if (!isdigit(argv[i + 1][0]))
            {
                std::cerr << "Invalid input" << std::endl;
                return 1; 
            }
            else
                num_steps = atoi(argv[i + 1]);
        }
        else 
        {
            std::cerr << "Unknown option: " << argv[i] << std::endl;
            return 1;
        }
    }

    //unsigned int seed = static_cast<unsigned int>(time(NULL));
    float avg_dist1 = 0.0;
    float avg_dist2 = 0.0;
    float avg_dist3 = 0.0;

    std::cout << "Number of walkers: " << num_walkers << "\n";
    std::cout << "Number of steps: " << num_steps << "\n"; 

    //kernel dimensions
    int block_size = 256;
    int grid_size = ((num_walkers + block_size) / block_size);

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    //-----------------------------------------------function 1-------------------------------------------------------------------------------------
    uint64_t* pageable_pos_x; //host memory
    uint64_t* pageable_pos_y; //host memory
    uint64_t* d_x; //device memory
    uint64_t* d_y; //device memory

    hipEventRecord(startEvent, 0);

    pageable_pos_x = (uint64_t*)malloc(sizeof(uint64_t) * num_walkers); //Allocate host memory, pageable
    pageable_pos_y = (uint64_t*)malloc(sizeof(uint64_t) * num_walkers); //Allocate host memory, pageable
    
    memset(pageable_pos_x, 0, sizeof(uint64_t) * num_walkers);
    memset(pageable_pos_y, 0, sizeof(uint64_t) * num_walkers);
    
    hipMalloc((uint64_t**)&d_x, sizeof(uint64_t) * num_walkers); //Allocate device memory
    hipMalloc((uint64_t**)&d_y, sizeof(uint64_t) * num_walkers);

    RandomWalk <<<grid_size, block_size >>> (d_x, d_y, num_walkers, num_steps, time(NULL)); //Execute kernel

    hipMemcpy(pageable_pos_x, d_x, sizeof(uint64_t) * num_walkers, hipMemcpyDeviceToHost); //Transfer data back to host memory
    hipMemcpy(pageable_pos_y, d_y, sizeof(uint64_t) * num_walkers, hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < num_walkers; ++i)
    {
        avg_dist1 += sqrt(pageable_pos_x[i] * pageable_pos_x[i] + pageable_pos_y[i] * pageable_pos_y[i]);
    }
    avg_dist1 /= num_walkers; 
    
    hipFree(pageable_pos_x);
    hipFree(pageable_pos_y);
    hipFree(d_x);
    hipFree(d_y);
    
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent); 
    hipEventElapsedTime(&elapsed_time, startEvent, stopEvent);

    std::cout << "Normal CUDA memory allocation:\n";
    std::cout << "    Time to calculate(microsec): " << elapsed_time*1000.0 << "\n";
    std::cout << "    Average distance from origin: " << avg_dist1 << "\n";
   
    //-----------------------------------------------function 2------------------------------------------------------------------------------------- 
    
    uint64_t* pinned_pos_x;
    uint64_t* pinned_pos_y;
    uint64_t* d2_x; //device memory
    uint64_t* d2_y; //device memory
    
    hipEventRecord(startEvent, 0);

    hipMalloc((uint64_t**)&d2_x, sizeof(uint64_t) * num_walkers); //Allocate device memory
    hipMalloc((uint64_t**)&d2_y, sizeof(uint64_t) * num_walkers);

    hipHostMalloc((void**)&pinned_pos_x, sizeof(uint64_t) * num_walkers); //host, pinned
    hipHostMalloc((void**)&pinned_pos_y, sizeof(uint64_t) * num_walkers);

    memset(pinned_pos_x, 0, sizeof(uint64_t) * num_walkers);
    memset(pinned_pos_y, 0, sizeof(uint64_t) * num_walkers);

    RandomWalk << <grid_size, block_size >> > (d2_x, d2_y, num_walkers, num_steps, time(NULL)); //Execute kernel

    hipMemcpy(pinned_pos_x, d2_x, sizeof(uint64_t) * num_walkers, hipMemcpyDeviceToHost); //Transfer data back to host memory
    hipMemcpy(pinned_pos_y, d2_y, sizeof(uint64_t) * num_walkers, hipMemcpyDeviceToHost);
    
    for (unsigned int i = 0; i < num_walkers; ++i)
        avg_dist2 += sqrt(pinned_pos_x[i] * pinned_pos_x[i] + pinned_pos_y[i]  * pinned_pos_y[i]);
    avg_dist2 /= num_walkers;

    hipHostFree(pinned_pos_x);
    hipHostFree(pinned_pos_y);
    hipFree(d2_x);
    hipFree(d2_y);
   
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&elapsed_time, startEvent, stopEvent);

    std::cout << "Pinned CUDA memory Allocation:\n";
    std::cout << "    Time to calculate(microsec): " << elapsed_time * 1000.0 << "\n";
    std::cout << "    Average distance from origin: " << avg_dist2 << "\n";
    
    //-----------------------------------------------function 3-------------------------------------------------------------------------------------
    uint64_t* m_positions_x; //managed memory
    uint64_t* m_positions_y;
    
    hipEventRecord(startEvent, 0);

    hipMallocManaged((void**)&m_positions_x, sizeof(uint64_t) * num_walkers);
    hipMallocManaged((void**)&m_positions_y, sizeof(uint64_t) * num_walkers);

    RandomWalk <<<grid_size, block_size >>> (m_positions_x, m_positions_y, num_walkers, num_steps, time(NULL));
    hipDeviceSynchronize();

    for (unsigned int i = 0; i < num_walkers; ++i)
        avg_dist3 += sqrt(m_positions_x[i] * m_positions_x[i] + m_positions_y[i] * m_positions_y[i]);
    avg_dist3 /= num_walkers;

    hipHostFree(m_positions_x);
    hipHostFree(m_positions_y);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&elapsed_time, startEvent, stopEvent);

    std::cout << "Managed CUDA memory allocation:\n";
    std::cout << "    Time to calculate(microsec): " << elapsed_time * 1000.0 << "\n";
    std::cout << "    Average distance from origin: " << avg_dist3 << "\n";
    std::cout << "Bye!" << "\n";

    //clean up
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    
    return 0;

}

